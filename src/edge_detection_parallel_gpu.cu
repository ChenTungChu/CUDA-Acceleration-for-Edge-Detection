/**
 * @file edge_detection_parallel_gpu.cu
 * @author jz544 & cc2396
 * @brief 
 * @version 1.0
 * @date 2022-05-01
 * 
 * compile: nvcc -o edge_detection_parallel_gpu edge_detection_parallel_gpu.cu -lm
 * run: ./edge_detection_parallel_gpu
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <locale>
// #include <cublas_v2.h>
// #include <cusolverDn.h>
#include <math.h>

#define IMG_PATH "./lena.pgm"
#define SAVE_PATH "./"
#define NUM_IMGS 100

typedef struct {
	char version[3]; 
	int width;
	int height;
	int maxGrayLevel;
	int **imageData;
	int **gx;
	int **gy;
} pgm;

void read_image(char* dir, pgm* image);   // used for reading image
void read_comments(FILE *input_image);    // used for reading image
int isspace(int argument); 			      // used for reading image
void padding(pgm* image);                 // padding 
void init_pgm_image(pgm* out, pgm image); // init pgm image for output
void min_max_normalization(pgm* image, int** matrix); // min max normalization
int convolution(pgm* image, int kernel[3][3], int row, int col);
void write_pgm_file(pgm* image, char* dir, int** matrix);

__global__ void sobel_edge_filter_cuda(int* dataIn, int* dataOut, int imgHeight,  int imgWidth) {
    int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    int index = yIndex * imgWidth + xIndex;
    int Gx = 0;
    int Gy = 0;
    if (xIndex > 0 && xIndex < imgWidth - 1 && yIndex > 0 && yIndex < imgHeight - 1)
    {
        Gx = dataIn[(yIndex - 1) * imgWidth + xIndex + 1] + 2 * dataIn[yIndex * imgWidth +  xIndex + 1] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]
            - (dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[yIndex * imgWidth  + xIndex - 1] + dataIn[(yIndex + 1) * imgWidth + xIndex - 1]);
        
        Gy = dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex - 1) *  imgWidth + xIndex] + dataIn[(yIndex - 1) * imgWidth + xIndex + 1]
            - (dataIn[(yIndex + 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex + 1) *  imgWidth + xIndex] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]);
        dataOut[index] = (abs(Gx) + abs(Gy)) / 2;
    }
}

int main(int argc, char* argv[]) {
    pgm image, out_image;   // define the input image and output image
    int num_of_imgs;
    dim3 grid(10,10);
    dim3 block(30,30);
    // number of threads = 30 * 30 * 30 * 30 = 900 * 900 = 810000
	if (argc == 1) {
		printf("[INFO] no arguments, use default number of images\n");
		printf("[INFO] number of images: %s\n", argv[1]);
		num_of_imgs = NUM_IMGS;
	}
	else {
		printf("[INFO] get num of images from command line\n");
		printf("[INFO] number of images: %s\n", argv[1]);
		num_of_imgs = atoi(argv[1]);
	}

    // read image 
    printf("[INFO] test image from %s\n", IMG_PATH);
    read_image(IMG_PATH, &image);
    padding(&image);
    init_pgm_image(&out_image, image);
    // allocate memory on device for input and output image'
    int *device_input_img, *device_output_img;
    int *host_flatten_input_img, *host_flatten_output_img;
    // convert input image into 1D array
    host_flatten_input_img = (int*)malloc(sizeof(int) * image.width * image.height);
    host_flatten_output_img = (int*)malloc(sizeof(int) * out_image.width * out_image.height);
    for (int i = 0; i < image.height; i++) {
        for (int j = 0; j < image.width; j++) {
            host_flatten_input_img[i * image.width + j] = image.imageData[i][j];
        }
    }
    hipMalloc((void**)&device_input_img, image.width * image.height * sizeof(int));
    hipMalloc((void**)&device_output_img, out_image.width * out_image.height * sizeof(int));
    struct timeval start, end;   // start and stop timer
    float el_time;               // elapsed time
    gettimeofday(&start, NULL);  // start counting time
    // kernel
	for (int i = 0; i < num_of_imgs; i++) {
        sobel_edge_filter_cuda<<<grid, block>>>(device_input_img, device_output_img, image.height, image.width);
	}
    hipDeviceSynchronize();
	// kernel
    gettimeofday(&end, NULL);   // stop counting time
    el_time = ((end.tv_sec - start.tv_sec) * 1000000u + end.tv_usec - start.tv_usec) / 1.e6;
    // copy the output image from device to host
    hipMemcpy(host_flatten_output_img, device_output_img, out_image.width * out_image.height * sizeof(int), hipMemcpyDeviceToHost);
    // convert output image into 2D array
    for (int i = 0; i < out_image.height; i++) {
        for (int j = 0; j < out_image.width; j++) {
            out_image.imageData[i][j] = host_flatten_output_img[i * out_image.width + j];
        }
    }
    min_max_normalization(&out_image, out_image.imageData);
	min_max_normalization(&out_image, out_image.gx);
	min_max_normalization(&out_image, out_image.gy);
	write_pgm_file(&out_image, "./gradient.pgm", out_image.imageData);
	write_pgm_file(&out_image, "./gradient_X.pgm", out_image.gx);
	write_pgm_file(&out_image, "./gradient_Y.pgm", out_image.gy);
	printf("[INFO] results has been saved in current directory.\n");
	printf("[INFO] time consumption: %e s\n", el_time);
	free(image.imageData);
	free(out_image.imageData);
	free(out_image.gx);
	free(out_image.gy);
    return 0;
}

void read_comments(FILE *input_image) {
	char ch;
	char line[100];

	while ((ch = fgetc(input_image)) != EOF && (isspace(ch)))  {
		;
    }
	if (ch == '#') {
        fgets(line, sizeof(line), input_image);
    } 
	else {
		fseek(input_image, -2L, SEEK_CUR);
	}
}

void read_image(char* dir, pgm* image) {
	FILE* input_image; 
	int i, j, num;

	input_image = fopen(dir, "rb");
	if (input_image == NULL) {
		printf("[ERROR] given figure is not found:%s\n", dir);
		return;
	} 
	
	fgets(image->version, sizeof(image->version), input_image);
	read_comments(input_image);

	fscanf(input_image, "%d %d %d", &image->width, &image->height, &image->maxGrayLevel);
	
	image->imageData = (int**) calloc(image->height, sizeof(int*));
	for(i = 0; i < image->height; i++) {
		image->imageData[i] = (int*) calloc(image->width, sizeof(int));
	}
	
	if (strcmp(image->version, "P2") == 0) {
		for (i = 0; i < image->height; i++) {
			for (j = 0; j < image->width; j++) {
				fscanf(input_image, "%d", &num);
				image->imageData[i][j] = num;
			}
		}	
	}
	else if (strcmp(image->version, "P5") == 0) {
		char *buffer;
		int buffer_size = image->height * image->width;
		buffer = (char*) malloc( ( buffer_size + 1) * sizeof(char));
		
		if(buffer == NULL) {
			printf("Can not allocate memory for buffer! \n");
			return;
		}
		fread(buffer, sizeof(char), image->width * image-> height, input_image);
		for (i = 0; i < image->height * image ->width; i++) {
			image->imageData[i / (image->width)][i % image->width] = buffer[i];
		}
		free(buffer);
	}
	fclose(input_image);
	printf("[INFO] pgm version: %s \tWidth: %d \tHeight: %d \tMaximum Gray Level: %d \n", image->version, image->width, image->height, image->maxGrayLevel);
}

void padding(pgm* image) {
	int i;
	for (i = 0; i < image->width; i++) {
		image->imageData[0][i] = 0;
		image->imageData[image->height - 1][i] = 0;
	}
	
	for (i = 0; i < image->height; i++) {
		image->imageData[i][0] = 0;
		image->imageData[i][image->width - 1] = 0;
	} 
}

void init_pgm_image(pgm* out, pgm image) {
	int i, j;
	strcpy(out->version, image.version);
	out->width = image.width;
	out->height = image.height;
	out->maxGrayLevel = image.maxGrayLevel;
	
	out->imageData = (int**) calloc(out->height, sizeof(int*));
	for(i = 0; i < out->height; i++) {
		out->imageData[i] = (int*)calloc(out->width, sizeof(int));
	}
	
	out->gx = (int**) calloc(out->height, sizeof(int*));
	for(i = 0; i < out->height; i++) {
		out->gx[i] = (int*)calloc(out->width, sizeof(int));
	}
	
	out->gy = (int**) calloc(out->height, sizeof(int*));
	for(i = 0; i < out->height; i++) {
		out->gy[i] = (int*)calloc(out->width, sizeof(int));
	}
	
	for(i = 0; i < out->height; i++) {
		for(j = 0; j < out->width; j++) {
			out->imageData[i][j] = image.imageData[i][j];
			out->gx[i][j] = image.imageData[i][j];
			out->gy[i][j] = image.imageData[i][j];
		};
	}
}

int convolution(pgm* image, int kernel[3][3], int row, int col) {
	int i, j, sum = 0;
	for (i = 0; i < 3; i++) {
		for (j = 0; j < 3; j++) {
			sum += image->imageData[i + row][j + col] * kernel[i][j];
		}
	}
	return sum;
}

void min_max_normalization(pgm* image, int** matrix) {
	int min = 1000000, max = 0, i, j;
	
	for(i = 0; i < image->height; i++) {
		for(j = 0; j < image->width; j++) {
			if (matrix[i][j] < min) {
				min = matrix[i][j];
			}
			else if (matrix[i][j] > max) {
				max = matrix[i][j];
			}
		}
	}
	
	for(i = 0; i < image->height; i++) {
		for(j = 0; j < image->width; j++) {
			double ratio = (double) (matrix[i][j] - min) / (max - min);
			matrix[i][j] = ratio * 255;
		}
	} 
}

void write_pgm_file(pgm* image, char* dir, int** matrix) {
	FILE* out_image;
	int i, j, count = 0;
	
	out_image = fopen(dir, "wb");
	fprintf(out_image, "%s\n", image->version);
	fprintf(out_image, "%d %d\n", image->width, image->height);
	fprintf(out_image, "%d\n", image->maxGrayLevel);
	
	if (strcmp(image->version, "P2") == 0) {
		for(i = 0; i < image->height; i++) {
			for(j = 0; j < image->width; j++) {
				fprintf(out_image,"%d", matrix[i][j]);
				if (count % 17 == 0) 
					fprintf(out_image,"\n");
				else 
					fprintf(out_image," ");
				count ++;
			}
		} 
	}
	else if (strcmp(image->version, "P5") == 0) {
		for(i = 0; i < image->height; i++) {
			for(j = 0; j < image->width; j++) {
				char num = matrix[i][j];
				fprintf(out_image,"%c", num);
			}
		} 
	} 
	fclose(out_image);
}